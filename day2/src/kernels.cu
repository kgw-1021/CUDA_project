#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "kernels.h"

__global__ void dummyKernel() { 
    printf("Block %d, Thread %d\n", blockIdx.x, threadIdx.x);
}