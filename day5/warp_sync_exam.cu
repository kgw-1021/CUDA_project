﻿#include <hip/hip_runtime.h>
#include <stdio.h>

int N = 128;  // 총 배열 크기
#define WARP_SIZE 32  // 워프 크기

__global__ void warpSyncKernel(float *A, float *B, float *C, int N) {
    __shared__ float sharedA[WARP_SIZE];
    __shared__ float sharedB[WARP_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpIdx = threadIdx.x % WARP_SIZE;

    if (idx < N) {
        sharedA[warpIdx] = A[idx];
        sharedB[warpIdx] = B[idx];
        __syncthreads(); // 블록 내 스레드 동기화

        C[idx] = sharedA[warpIdx] + sharedB[warpIdx];
    }
}

int main() {
    size_t size = N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // 호스트 배열 초기화
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // 디바이스 메모리 할당
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 호스트에서 디바이스로 데이터 복사
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 스레드와 블록 설정 (1D 배열이므로 1차원 그리드 설정)
    int threadsPerBlock = WARP_SIZE;
    int blocksPerGrid = (N + WARP_SIZE - 1) / WARP_SIZE;

    // 커널 실행
    warpSyncKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 결과를 호스트로 복사
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 결과 검증
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != 3.0f) {  // 각 요소가 1 + 2 = 3이어야 함
            printf("Error at index %d: %f != 3.0\n", i, h_C[i]);
            success = false;
            break;
        }
    }

    if (success) {
        printf("Kernel execution successful!\n");
    }

    // 메모리 해제
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
